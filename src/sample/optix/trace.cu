#include "hip/hip_runtime.h"
#include "shared.h"
#include <optix.h>

extern "C" {
    __constant__ LaunchParameters launchParameters;
}

extern "C"
__global__ void __raygen__test1() {
    auto launchIndex = optixGetLaunchIndex();
    launchParameters.image[launchIndex.y * launchParameters.width + launchIndex.x] = make_uchar4(255, 0, 0, 255); // clear to pure red.
}
