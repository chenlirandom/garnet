#include "hip/hip_runtime.h"
#include "shared.h"
#include <optix.h>
#include <stdio.h>

extern "C" {
    __constant__ LaunchParameters launchParameters;
}

extern "C"
__global__ void __raygen__test1() {
    auto launchIndex = optixGetLaunchIndex();
    launchParameters.image[launchIndex.y * launchParameters.width + launchIndex.x] = make_uchar4(
        (uint8_t)( launchIndex.x * 255 / ( launchParameters.width  - 1 ) ),
        (uint8_t)( launchIndex.y * 255 / ( launchParameters.height - 1 ) ),
        0,
        255
    );
}
